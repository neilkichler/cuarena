#include "hip/hip_runtime.h"
#include <cuarena/cuarena.hpp>

#include <cstdio>

#define CU_CHECK(x)                                                                               \
    do {                                                                                          \
        hipError_t err = x;                                                                         \
        if (err != hipSuccess) {                                                                \
            const char *error_string;                                                             \
            if (hipDrvGetErrorString(err, &error_string) == hipErrorInvalidValue) {               \
                error_string = "Unknown error";                                                   \
            }                                                                                     \
            const char *error_name;                                                               \
            if (hipDrvGetErrorName(err, &error_name) == hipErrorInvalidValue) {                   \
                error_name = "hipErrorUnknown";                                                \
            }                                                                                     \
            fprintf(stderr, "CUDA Driver API error in %s at %s:%d: %s (%s = %d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, error_string, error_name, err);                           \
            abort();                                                                              \
        }                                                                                         \
    } while (0)

__global__ void kernel(int *xs) { xs[0] = 42; }

int main()
{
    using namespace cu;
    hipSetDevice(0);

    constexpr int n = 1024;
    int h_xs[n];

    auto n_bytes = sizeof(int) * n;

    arena a;
    memblk buffer = a.allocate(n_bytes);
    int *d_xs     = new (buffer.data()) int[n];

    kernel<<<1, 1>>>(d_xs);
    hipMemcpy(h_xs, d_xs, n_bytes, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("h_xs[0] = %d\n", h_xs[0]);
    return 0;
}
