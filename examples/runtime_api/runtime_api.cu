#include "hip/hip_runtime.h"
#include <cuarena/cuarena.hpp>

#include <cstdio>

#define CUDA_CHECK(x)                                                                \
    do {                                                                             \
        hipError_t err = x;                                                         \
        if (err != hipSuccess) {                                                    \
            fprintf(stderr, "CUDA error in %s at %s:%d: %s (%s=%d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, hipGetErrorString(err),                     \
                    hipGetErrorName(err), err);                                     \
            abort();                                                                 \
        }                                                                            \
    } while (0)

__global__ void kernel(int *xs) { xs[0] = 42; }

int main()
{
    using namespace cu;
    CUDA_CHECK(hipSetDevice(0));

    constexpr int n = 1024;
    int h_xs[n];

    auto n_bytes = sizeof(int) * n;

    arena a;
    memblk buffer = a.allocate(n_bytes);
    int *d_xs     = new (buffer.data()) int[n];

    kernel<<<1, 1>>>(d_xs);
    CUDA_CHECK(hipMemcpy(h_xs, d_xs, n_bytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize());
    printf("h_xs[0] = %d\n", h_xs[0]);
    return 0;
}
