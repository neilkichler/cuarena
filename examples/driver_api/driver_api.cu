#include "hip/hip_runtime.h"
#include <cuarena/cuarena.hpp>

#include <cstdio>

#define CU_CHECK(x)                                                                               \
    do {                                                                                          \
        hipError_t err = x;                                                                         \
        if (err != hipSuccess) {                                                                \
            const char *error_string;                                                             \
            if (hipDrvGetErrorString(err, &error_string) == hipErrorInvalidValue) {               \
                error_string = "Unknown error";                                                   \
            }                                                                                     \
            const char *error_name;                                                               \
            if (hipDrvGetErrorName(err, &error_name) == hipErrorInvalidValue) {                   \
                error_name = "hipErrorUnknown";                                                \
            }                                                                                     \
            fprintf(stderr, "CUDA Driver API error in %s at %s:%d: %s (%s = %d)\n", __FUNCTION__, \
                    __FILE__, __LINE__, error_string, error_name, err);                           \
            abort();                                                                              \
        }                                                                                         \
    } while (0)

__global__ void kernel(int *xs) { xs[0] = 42; }

int main()
{
    using namespace cu;

    hipCtx_t ctx;
    hipDevice_t dev;

    CU_CHECK(hipInit(0));
    CU_CHECK(hipDevicePrimaryCtxRetain(&ctx, 0));
    CU_CHECK(hipCtxSetCurrent(ctx));
    CU_CHECK(hipCtxGetDevice(&dev));

    constexpr int n = 1024;

    int h_xs[n];

    auto n_bytes = sizeof(int) * n;
    arena a(ctx);
    memblk buffer = a.allocate(n_bytes);
    int *d_xs     = new (buffer.data()) int[n];

    kernel<<<1, 1>>>(d_xs);
    CU_CHECK(hipMemcpyDtoH(h_xs, (hipDeviceptr_t) d_xs, n_bytes));
    CU_CHECK(hipCtxSynchronize());

    printf("xs[0] = %d\n", h_xs[0]);

    CU_CHECK(hipDevicePrimaryCtxRelease(0));

    return 0;
}
