#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <chrono>
#include <thread>
#include <assert.h>

#include "cuvector.cuh"

typedef std::chrono::nanoseconds ReportingDuration;

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

template<typename V>
void measureGrow(V& v, size_t minN, size_t maxN, std::vector<ReportingDuration>& durations)
{
    for (size_t n = minN; n <= maxN; n <<= 1) {
        typedef std::chrono::time_point<std::chrono::steady_clock> time_point;

        time_point start = std::chrono::steady_clock::now();
        hipError_t status = v.grow(n);
        time_point end = std::chrono::steady_clock::now();

        durations.push_back(std::chrono::duration_cast<ReportingDuration>(end - start));
        // In non-release, verify the memory is accessible and everything worked properly
        assert(hipSuccess == status);
        assert(hipSuccess == hipMemsetD8((hipDeviceptr_t)v.getPointer(), 0, v.getSize()));
        assert(hipSuccess == hipCtxSynchronize());
    }
}

template<typename Allocator, typename Elem>
void runVectorPerfTest(hipCtx_t ctx, size_t minN, size_t maxN,
                       std::vector<ReportingDuration>& noReserveDurations,
                       std::vector<ReportingDuration>& reserveDurations)
{
    typedef cuda_utils::Vector<Elem, Allocator> VectorDUT;

    if (false) {
        // Warm-up
        VectorDUT dut(ctx);
        if (!dut.grow(maxN)) {
            std::cerr << "Failed to grow to max elements, test invalid!\n" << std::endl;
            return;
        }
    }

    // Wait for the OS to settle it's GPU pages from past perf runs
    std::this_thread::sleep_for(std::chrono::seconds(2));
    {
        // Measure without reserving
        VectorDUT dut(ctx);
        measureGrow(dut, minN, maxN, noReserveDurations);
    }

    // Wait for the OS to settle it's GPU pages from past perf runs
    std::this_thread::sleep_for(std::chrono::seconds(2));
    {
        size_t free = 0ULL;
        VectorDUT dut(ctx);

        dut.reserve(maxN);
        CHECK_DRV(hipMemGetInfo(&free, NULL));
        std::cout << "\tReserved " << maxN << " elements..." << std::endl
                  << "\tFree Memory: " << (float)free / std::giga::num << "GB" << std::endl;

        measureGrow(dut, minN, maxN, reserveDurations);
    }
}

int main()
{
    size_t free;
    typedef unsigned char ElemType;
    hipCtx_t ctx;
    hipDevice_t dev;
    int supportsVMM = 0;

    CHECK_DRV(hipInit(0));
    CHECK_DRV(hipDevicePrimaryCtxRetain(&ctx, 0));
    CHECK_DRV(hipCtxSetCurrent(ctx));
    CHECK_DRV(hipCtxGetDevice(&dev));

    std::vector<std::vector<ReportingDuration> > durations(4);

    CHECK_DRV(hipMemGetInfo(&free, NULL));

    std::cout << "Total Free Memory: " << (float)free / std::giga::num << "GB" << std::endl;

    // Skip the smaller cases
    const size_t minN = (2ULL * 1024ULL * 1024ULL + sizeof(ElemType) - 1ULL) / sizeof(ElemType);
    // Use at max about 75% of all vidmem for perf testing
    // Also, some vector allocators like MemAlloc cannot handle more than this,
    // as they would run out of memory during the grow algorithm
    const size_t maxN = 3ULL * free / (4ULL * sizeof(ElemType));

    std::cout << "====== hipMalloc ElemSz=" << sizeof(ElemType) << " ======" << std::endl;
    runVectorPerfTest<cuda_utils::VectorMemAlloc, ElemType>(ctx, minN, maxN, durations[0], durations[1]);
    // std::cout << "====== hipMallocManaged ElemSz=" << sizeof(ElemType) << " ======" << std::endl;
    // runVectorPerfTest<cuda_utils::VectorMemAllocManaged, ElemType>(ctx, minN, maxN, durations[2], durations[3]);
    // durations[2].resize(maxN);
    // durations[3].resize(maxN);

    CHECK_DRV(hipDeviceGetAttribute(&supportsVMM, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, dev));

    if (supportsVMM) {
        // durations.resize(durations.size() + 2);
        std::cout << "====== hipMemMap ElemSz=" << sizeof(ElemType) << " ======" << std::endl;
        runVectorPerfTest<cuda_utils::VectorMemMap, ElemType>(ctx, minN, maxN, durations[2], durations[3]);
    }

    // Quick and dirty table of results
    std::cout << "Size(bytes)    | "
              << "Alloc(us)      | "
              << "AllocRes(us)   | ";
              // << "Managed(us)    | "
              // << "ManagedRes(us) | ";

    if (supportsVMM) {
        std::cout << "hipMemMap(us)   | "
                  << "cuMemMapRes(us)| ";
    }

    std::cout << std::endl;

    for (size_t i = 0; i < durations[0].size(); i++) {
        std::cout << std::left << std::setw(15) << std::setfill(' ') << (minN << i) << "| ";
        for (size_t j = 0; j < durations.size(); j++) {
            std::cout << std::left << std::setw(15) << std::setfill(' ')
                      << std::setprecision(2) << std::fixed
                      << std::chrono::duration_cast <std::chrono::duration<float, std::micro> >(durations[j][i]).count() << "| ";
        }
        std::cout << std::endl;
    }

    CHECK_DRV(hipDevicePrimaryCtxRelease(0));

    return 0;
}
