#include "cuarena.cuh"

struct dummy
{
    int a;
    int b;
    int c;
    int d;
};

int main(int argc, char *argv[])
{
    hipCtx_t ctx;
    hipDevice_t dev;

    CU_CHECK(hipInit(0));
    CU_CHECK(hipDevicePrimaryCtxRetain(&ctx, 0));
    CU_CHECK(hipCtxSetCurrent(ctx));
    CU_CHECK(hipCtxGetDevice(&dev));

    int supports_virtual_memory = 0;
    CU_CHECK(hipDeviceGetAttribute(&supports_virtual_memory, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, dev));
    if (supports_virtual_memory) {
        printf("Virtual memory support detected\n");
    }

    printf("Context: %p, Device: %d\n", ctx, dev);

    auto n_bytes = 2_GB;
    arena a(ctx);
    memblk buffer = a.allocate(n_bytes);
    a.deallocate(buffer);
    memblk buffer2 = a.allocate(n_bytes);
    memblk buffer3 = a.allocate(n_bytes);
    memblk buffer4 = a.allocate(n_bytes);

    // pool p(ctx, { .slot_size = sizeof(dummy), .capacity = 4 * sizeof(dummy), .alignment = 16 });
    //
    // memblk b1 = p.allocate();
    // memblk b2 = p.allocate();
    // memblk b3 = p.allocate();
    // p.deallocate(b3);
    // memblk b4 = p.allocate();
    // memblk b5 = p.allocate();
    // memblk b6 = p.allocate();
    // p.clear();

    CU_CHECK(hipDevicePrimaryCtxRelease(0));

    return 0;
}
