#include "cuarena.cuh"

int main(int argc, char *argv[])
{
    hipCtx_t ctx;
    hipDevice_t dev;

    CU_CHECK(hipInit(0));
    CU_CHECK(hipDevicePrimaryCtxRetain(&ctx, 0));
    CU_CHECK(hipCtxSetCurrent(ctx));
    CU_CHECK(hipCtxGetDevice(&dev));

    int supports_virtual_memory = 0;
    CU_CHECK(hipDeviceGetAttribute(&supports_virtual_memory, CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED, dev));
    if (supports_virtual_memory) {
        printf("Virtual memory support detected\n");
    }

    printf("Context: %p, Device: %d\n", ctx, dev);

    auto n_bytes = 2_GB;
    arena a(ctx);
    memblk buffer = a.allocate(n_bytes);
    a.deallocate(buffer);
    memblk buffer2 = a.allocate(n_bytes);
    memblk buffer3 = a.allocate(n_bytes);
    memblk buffer4 = a.allocate(n_bytes);

    CU_CHECK(hipDevicePrimaryCtxRelease(0));

    return 0;
}
